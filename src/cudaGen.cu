#include "hip/hip_runtime.h"
#include "cudaGen.h"

__global__ void cuda::countBinary(int n, std::bitset<BITSET_SIZE> *results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (1 << n)) {
        results[idx] = std::bitset<BITSET_SIZE>(idx);
    }
}

int cuda::generate() {
    int n;
    std::cout << "Enter the number of binary digits (n): ";
    std::cin >> n;

    if (n > BITSET_SIZE) {
        std::cerr << "Error: n must be less than or equal to " << BITSET_SIZE << std::endl;
        return 1;
    }

    int maxNumber = (1 << n); // 2^n
    std::vector<std::bitset<BITSET_SIZE>> bitsets(maxNumber);

    std::bitset<BITSET_SIZE> *d_results;
    hipMalloc((void**)&d_results, maxNumber * sizeof(std::bitset<BITSET_SIZE>));

    int threadsPerBlock = 256;
    int blocksPerGrid = (maxNumber + threadsPerBlock - 1) / threadsPerBlock;

    auto start = std::chrono::high_resolution_clock::now();

    countBinary<<<blocksPerGrid, threadsPerBlock>>>(n, d_results);
    hipDeviceSynchronize();
    hipMemcpy(bitsets.data(), d_results, maxNumber * sizeof(std::bitset<BITSET_SIZE>), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    std::cout << "Time taken: " << duration.count() << " milliseconds" << std::endl;
    std::cout << "Last generated number is: "<< bitsets[maxNumber - 1].to_string() << std::endl;
 
    hipFree(d_results);
    return 0;
}
