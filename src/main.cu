#include "cheeseOps.h"

int main(){
    cheese::Cheese cheese = cheese::MakeCheese(2,2,2);
    auto cheesePtr = std::make_shared<cheese::Cheese>(cheese);
    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheesePtr);

    int n;
    std::cout << "Enter the number of binary digits (n): ";
    std::cin >> n;
    auto allTheCheeses = cuda::cheesenerate<8>(n);

    for (const auto &holeset: allTheCheeses) {
        cheese::Cheese cheeseCopy = cheese;
        std::cout<<"{"<<holeset.to_string()<<"}"<<std::endl;
        cheeseCopy.info();
        keeper->carveTheCheese<8>(cheeseCopy, holeset);
        std::cout<<"============================="<<std::endl;
        keeper->verifyCheese(cheeseCopy, 3, 5, 2, false);
        cheeseCopy.info();
    }
   
    return 0;
}