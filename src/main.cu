#include "cheeseOps.h"

int main(){
    std::shared_ptr<cheese::Cheese> cheese = cheese::MakeCheese(2,2,2);
    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheese);

    /*  
    bool test = keeper->verifyCheese(cheese);
    cheese->info();
    for (const auto &cheeseball: cheese->cheeseBalls) {
        if(keeper->validateCheeseBall(cheeseball, cheese, 3, 5, 2)) std::cout<<"CheeseBall "<<cheeseball->index<<" is valid!"<<std::endl;
        else std::cout<<"CheeseBall "<<cheeseball->index<<" is not valid :(("<<std::endl; 
    }
    */

    cuda::generate();
}