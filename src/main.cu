#include "hip/hip_runtime.h"
#include "cheeseOps.h"

#define CHEESEBALLS 16 

int main(){
    cheese::Cheese cheese = cheese::MakeCheese(2,4,2);
    auto cheesePtr = std::make_shared<cheese::Cheese>(cheese);
    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheesePtr);

    auto allTheCheeses = cuda::cheesenerate<CHEESEBALLS>(CHEESEBALLS);
    size_t cheeseCount = allTheCheeses.size();

    int maxHoles = -1;
    int progress = 0;
    std::bitset<CHEESEBALLS> mostHolesomeCheese; 

    auto start = std::chrono::high_resolution_clock::now();
    #pragma omp parallel for reduction(max:maxHoles)
    for (const auto &holeset: allTheCheeses) {
        cheese::Cheese cheeseCopy = cheese;
        keeper->carveTheCheese<CHEESEBALLS>(cheeseCopy, holeset);

        #pragma omp atomic
        progress++;
        if (omp_get_thread_num() % 3 == 0 && progress % (cheeseCount / 100) == 0 || progress == cheeseCount) {
            std::cout << "\rCheesed: " << (100 * progress / cheeseCount) << "%" << std::flush;
        }

        if (keeper->verifyCheese(cheeseCopy, 3, 5, 2, false)) {
            int holes = holeset.size() - holeset.count();
            #pragma omp critical
            {
                if (holes > maxHoles) {
                    maxHoles = holes;
                    mostHolesomeCheese = holeset;
                }
            } 
        }
    }
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    std::cout<<std::endl;
    std::cout<<std::endl;
    std::cout << "Openmp took " << duration.count() << " milliseconds to find the solution" << std::endl;

    keeper->carveTheCheese<CHEESEBALLS>(cheese, mostHolesomeCheese);
    if (keeper->verifyCheese(cheese, 3, 5, 2, false)) std::cout<<"One of optimal cheeses is "<<mostHolesomeCheese.to_string()<<", number of holes: "<<CHEESEBALLS-mostHolesomeCheese.count()<<" out of "<<CHEESEBALLS<<" cheeseballs"<<std::endl;
    
    return 0;
}