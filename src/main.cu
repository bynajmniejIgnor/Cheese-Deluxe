#include "cheeseOps.h"

int main(){
    cheese::Cheese cheese = cheese::MakeCheese(2,2,2);
    auto cheesePtr = std::make_shared<cheese::Cheese>(cheese);
    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheesePtr);


    /*  
    bool test = keeper->verifyCheese(cheese);
    cheese->info();
    for (const auto &cheeseball: cheese->cheeseBalls) {
        if(keeper->validateCheeseBall(cheeseball, cheese, 3, 5, 2)) std::cout<<"CheeseBall "<<cheeseball->index<<" is valid!"<<std::endl;
        else std::cout<<"CheeseBall "<<cheeseball->index<<" is not valid :(("<<std::endl; 
    }
    */
    int n;
    std::cout << "Enter the number of binary digits (n): ";
    std::cin >> n;
    auto allTheCheeses = cuda::cheesenerate<8>(n);

    std::cout<<"Highest number generated: " << allTheCheeses[allTheCheeses.size()-1].to_string() <<std::endl;

    std::cout<<"====== ORIGINAL CHEESE ======="<<std::endl;
    cheese.info();
    std::cout<<"====== MODIFIED CHEESE ======="<<std::endl;
    keeper->carveTheCheese<8>(cheese, allTheCheeses[7]);
    cheese.info();
    return 0;
}