#include "cheeseOps.h"

int main(){
    std::shared_ptr<cheese::Cheese> cheese = cheese::MakeCheese(2,2,2);
    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheese);

    /*  
    bool test = keeper->verifyCheese(cheese);
    cheese->info();
    for (const auto &cheeseball: cheese->cheeseBalls) {
        if(keeper->validateCheeseBall(cheeseball, cheese, 3, 5, 2)) std::cout<<"CheeseBall "<<cheeseball->index<<" is valid!"<<std::endl;
        else std::cout<<"CheeseBall "<<cheeseball->index<<" is not valid :(("<<std::endl; 
    }
    */
    int n;
    std::cout << "Enter the number of binary digits (n): ";
    std::cin >> n;
    auto allTheCheeses = cuda::cheesenerate<32>(n);

    std::cout<<"Highest number generated: " << allTheCheeses[allTheCheeses.size()-1].to_string() <<std::endl;
    return 0;
}