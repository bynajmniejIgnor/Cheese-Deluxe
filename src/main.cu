#include "hip/hip_runtime.h"
#include "cheeseOps.h"

#define CHEESEBALLS 32 

int main(){
    cheese::Cheese cheese = cheese::MakeCheese(2,4,2);
    auto cheesePtr = std::make_shared<cheese::Cheese>(cheese);
    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheesePtr);

    int n;
    std::cout << "Enter the number of binary digits (n): ";
    std::cin >> n;
    auto allTheCheeses = cuda::cheesenerate<CHEESEBALLS>(n);

    int maxHoles = -1;
    std::bitset<CHEESEBALLS> mostHolesomeCheese; 

    #pragma omp parallel for reduction(max:maxHoles)
    for (const auto &holeset: allTheCheeses) {
        cheese::Cheese cheeseCopy = cheese;
        keeper->carveTheCheese<CHEESEBALLS>(cheeseCopy, holeset);
        if (keeper->verifyCheese(cheeseCopy, 3, 5, 2, false)) {
            int holes = holeset.size() - holeset.count();
            #pragma omp critical
            {
                if (holes > maxHoles) {
                    maxHoles = holes;
                    mostHolesomeCheese = holeset;
                }
            } 
        }
    }


    keeper->carveTheCheese<CHEESEBALLS>(cheese, mostHolesomeCheese);
    if (keeper->verifyCheese(cheese, 3, 5, 2, false)) std::cout<<"One of optimal cheeses is "<<mostHolesomeCheese<<", number of holes: "<<CHEESEBALLS-mostHolesomeCheese.count()<<std::endl;
    cheese.info();
    
    return 0;
}