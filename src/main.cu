#include "cudaGen.h"
#include "cheese.h"

int main(){
    //cuda::generate();
   std::shared_ptr<cheese::Cheese> cheese = cheese::MakeCheese(2,2,2);

   cheese->bindSlices();
   cheese->stackSlices();
   cheese->solidifyCheese();
   cheese->ageTheCheese();
   cheese->info();
}