#include "hip/hip_runtime.h"
#include "cheeseOps.h"

#define CHEESEBALLS 24 
#define CHEESE_WIDTH 3
#define CHEESE_LENGTH 4
#define CHEESE_HEIGHT 2

int main(){
    cheese::Cheese cheese = cheese::MakeCheese(CHEESE_WIDTH, CHEESE_LENGTH, CHEESE_HEIGHT);
    auto cheesePtr = std::make_shared<cheese::Cheese>(cheese);
    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheesePtr);

    auto allTheCheeses = cuda::cheesenerate<CHEESEBALLS>(CHEESEBALLS);
    size_t cheeseCount = allTheCheeses.size();

    int maxHoles = -1;
    int progress = 0;
    std::bitset<CHEESEBALLS> mostHolesomeCheese; 

    auto start = std::chrono::high_resolution_clock::now();
    #pragma omp parallel for reduction(max:maxHoles)
    for (const auto &holeset: allTheCheeses) {
        cheese::Cheese cheeseCopy = cheese;
        keeper->carveTheCheese<CHEESEBALLS>(cheeseCopy, holeset);

        #pragma omp atomic
        progress++;
        if (omp_get_thread_num() % 3 == 0 && progress % (cheeseCount / 100) == 0 || progress == cheeseCount) {
            std::cout << "\rCheesed: " << (100 * progress / cheeseCount) << "%" << std::flush;
        }

        if (keeper->verifyCheese(cheeseCopy, 3, 5, 2, false)) {
            int holes = holeset.size() - holeset.count();
            #pragma omp critical
            {
                if (holes > maxHoles) {
                    maxHoles = holes;
                    mostHolesomeCheese = holeset;
                }
            } 
        }
    }
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    std::cout<<std::endl;
    std::cout<<std::endl;
    std::cout << "Openmp took " << duration.count() << " milliseconds to find the solution" << std::endl;

    cheese::Cheese cheeseCopy = cheese;
    keeper->carveTheCheese<CHEESEBALLS>(cheeseCopy, mostHolesomeCheese);
    if (keeper->verifyCheese(cheeseCopy, 3, 5, 2, false)) std::cout<<"One of optimal cheeses is "<<mostHolesomeCheese.to_string()<<", number of holes: "<<CHEESEBALLS-mostHolesomeCheese.count()<<" out of "<<CHEESEBALLS<<" cheeseballs"<<std::endl;


    maxHoles = -1;
    progress = 0;
    std::cout<<"Now cheeseing sequentially..."<<std::endl;

    start = std::chrono::high_resolution_clock::now();
    for (const auto &holeset: allTheCheeses) {
        cheese::Cheese cheeseCopy = cheese;
        keeper->carveTheCheese<CHEESEBALLS>(cheeseCopy, holeset);
        progress++;
        if (progress % (cheeseCount / 100) == 0 || progress == cheeseCount) {
            std::cout << "\rCheesed: " << (100 * progress / cheeseCount) << "%" << std::flush;
        }

        if (keeper->verifyCheese(cheeseCopy, 3, 5, 2, false)) {
            int holes = holeset.size() - holeset.count();
            {
                if (holes > maxHoles) {
                    maxHoles = holes;
                    mostHolesomeCheese = holeset;
                }
            } 
        }
    }

    end = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    std::cout<<std::endl;
    std::cout<<std::endl;
    std::cout << "Sequential algorithm took " << duration.count() << " milliseconds to find the solution" << std::endl;

    cheeseCopy = cheese;
    keeper->carveTheCheese<CHEESEBALLS>(cheeseCopy, mostHolesomeCheese);
    if (keeper->verifyCheese(cheeseCopy, 3, 5, 2, false)) std::cout<<"One of optimal cheeses is "<<mostHolesomeCheese.to_string()<<", number of holes: "<<CHEESEBALLS-mostHolesomeCheese.count()<<" out of "<<CHEESEBALLS<<" cheeseballs"<<std::endl;
    
    return 0;
}