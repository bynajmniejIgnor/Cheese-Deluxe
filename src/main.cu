#include "cudaGen.h"
#include "cheeseOps.h"

int main(){
    //cuda::generate();
   std::shared_ptr<cheese::Cheese> cheese = cheese::MakeCheese(2,2,1);
   cheese->info();

    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheese);

    bool test = keeper->verify(cheese);
}