#include "hip/hip_runtime.h"
#include "cheeseOps.h"

#define CHEESEBALLS 16 

int main(){
    cheese::Cheese cheese = cheese::MakeCheese(2,2,2);
    auto cheesePtr = std::make_shared<cheese::Cheese>(cheese);
    std::shared_ptr<cheeseOps::CheeseKeeper> keeper = cheeseOps::EducateCheeseKeeper(cheesePtr);

    auto allTheCheeses = cuda::cheesenerate<CHEESEBALLS>(CHEESEBALLS);
    size_t cheeseCount = allTheCheeses.size();

    int maxHoles = -1;
    int progress = 0;
    int c;
    std::cin>>c;
    std::bitset<CHEESEBALLS> mostHolesomeCheese; 

    #pragma omp parallel for reduction(max:maxHoles)
    for (const auto &holeset: allTheCheeses) {
        cheese::Cheese cheeseCopy = cheese;
        keeper->carveTheCheese<CHEESEBALLS>(cheeseCopy, holeset);

        #pragma omp atomic
        progress++;
        if (omp_get_thread_num() % 3 == 0 && progress % (cheeseCount / 100) == 0 || progress == cheeseCount) {
            std::cout << "\rCheesed: " << (100 * progress / cheeseCount) << "%" << std::flush;
        }

        if (keeper->verifyCheese(cheeseCopy, 3, 5, 2, false)) {
            int holes = holeset.size() - holeset.count();
            #pragma omp critical
            {
                if (holes > maxHoles) {
                    maxHoles = holes;
                    mostHolesomeCheese = holeset;
                }
            } 
        }
    }

    std::cout<<std::endl;
    keeper->carveTheCheese<CHEESEBALLS>(cheese, mostHolesomeCheese);
    if (keeper->verifyCheese(cheese, 3, 5, 2, true)) std::cout<<"One of optimal cheeses is "<<mostHolesomeCheese.to_string()<<", number of holes: "<<CHEESEBALLS-mostHolesomeCheese.count()<<std::endl;
    cheese.info();
    
    return 0;
}