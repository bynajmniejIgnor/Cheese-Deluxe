#include <iostream>
#include <hip/hip_runtime.h>
#include <bitset>
#include <vector>
#include <chrono>

#define BITSET_SIZE 64 

__global__ void countBinary(int n, std::bitset<BITSET_SIZE> *results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (1 << n)) {
        results[idx] = std::bitset<BITSET_SIZE>(idx);
    }
}

int main() {
    int n;
    std::cout << "Enter the number of binary digits (n): ";
    std::cin >> n;

    if (n > BITSET_SIZE) {
        std::cerr << "Error: n must be less than or equal to BITSET_SIZE." << std::endl;
        return 1;
    }

    int maxNumber = (1 << n); // 2^n
    std::vector<std::bitset<BITSET_SIZE>> bitsets(maxNumber);

    std::bitset<BITSET_SIZE> *d_results;
    hipMalloc((void**)&d_results, maxNumber * sizeof(std::bitset<BITSET_SIZE>));

    int threadsPerBlock = 256;
    int blocksPerGrid = (maxNumber + threadsPerBlock - 1) / threadsPerBlock;


    auto start = std::chrono::high_resolution_clock::now();

    countBinary<<<blocksPerGrid, threadsPerBlock>>>(n, d_results);
    hipDeviceSynchronize();

    hipMemcpy(bitsets.data(), d_results, maxNumber * sizeof(std::bitset<BITSET_SIZE>), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    // Output the duration
    std::cout << "Time taken: " << duration.count() << " milliseconds" << std::endl;
    std::cout << "Last generated number is: "<< bitsets[maxNumber - 1].to_string() << std::endl;
 

    /*
    for (int i = 0; i < maxNumber; ++i) {
        std::cout << bitsets[i].to_string().substr(BITSET_SIZE - n) << std::endl;
    }
    */
    hipFree(d_results);
    return 0;
}
